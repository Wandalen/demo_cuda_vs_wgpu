#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAddition(float* a, float* b, float* result, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        result[index] = a[index] + b[index];
    }
}

__global__ void vectorSum(float* a, int size, float* sum) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float sharedSum;

    if (tid < size) {
        sharedSum = 0;

        atomicAdd(&sharedSum, a[tid]);

        __syncthreads();

        if (threadIdx.x == 0) {
            atomicAdd(sum, sharedSum);
        }
    }
}

__device__ double optimized_array_cuda(const double* arr, int start, int end) {
    if (end == start) {
        return arr[end];
    }
    if (end - start == 1) {
        return arr[start] + arr[end];
    }
    else {
        int mid = (end - start) / 2 + start;
        double sum1, sum2;
        sum1 = optimized_array_cuda(arr, start, mid);
        sum2 = optimized_array_cuda(arr, mid + 1, end);
        return sum1 + sum2;
    }
}

__global__ void kernel(double* result, const double* arr, int start, int end) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *result = optimized_array_cuda(arr, start, end);
    }
}

// int main() {
//     int size = 1000000;
//     int byteSize = size * sizeof(float);

//     float* hostVectorA = new float[size];
//     float* hostVectorB = new float[size];
//     float* hostResult = new float[size];

//     for (int i = 0; i < size; ++i) {
//         hostVectorA[i] = static_cast<float>(i);
//         hostVectorB[i] = static_cast<float>(i * 2);
//     }

//     float* deviceVectorA, * deviceVectorB, * deviceResult;
//     cudaMalloc((void**)&deviceVectorA, byteSize);
//     cudaMalloc((void**)&deviceVectorB, byteSize);
//     cudaMalloc((void**)&deviceResult, byteSize);

//     cudaMemcpy(deviceVectorA, hostVectorA, byteSize, cudaMemcpyHostToDevice);
//     cudaMemcpy(deviceVectorB, hostVectorB, byteSize, cudaMemcpyHostToDevice);

//     float* deviceSum;
//     cudaMalloc((void**)&deviceSum, sizeof(float));
//     cudaMemset(deviceSum, 0, sizeof(float));

//     int blockSize = 256;
//     int gridSize = (size + blockSize - 1) / blockSize;

//     cudaEvent_t start, stop;
//     cudaEventCreate(&start);
//     cudaEventCreate(&stop);

//     cudaEventRecord(start, 0);

//     // vectorAddition<<<gridSize, blockSize>>>(deviceVectorA, deviceVectorB, deviceResult, size);
//     vectorSum<<<gridSize, blockSize>>>(deviceVectorA, size, deviceSum);

//     cudaEventRecord(stop, 0);
//     cudaEventSynchronize(stop);

//     float elapsedTime;
//     cudaEventElapsedTime(&elapsedTime, start, stop);

//     cudaMemcpy(hostResult, deviceResult, byteSize, cudaMemcpyDeviceToHost);

//     // for (int i = 0; i < size; ++i) {
//     //     std::cout << hostVectorA[i] << " + " << hostVectorB[i] << " = " << hostResult[i] << std::endl;
//     // }

//     float hostSum;
//     cudaMemcpy(&hostSum, deviceSum, sizeof(float), cudaMemcpyDeviceToHost);

//     std::cout << "Sum of vector elements: " << hostSum << std::endl;
//     std::cout << "Elapsed Time: " << elapsedTime << " ms" << std::endl;

//     delete[] hostVectorA;
//     delete[] hostVectorB;
//     delete[] hostResult;
//     cudaFree(deviceVectorA);
//     cudaFree(deviceVectorB);
//     cudaFree(deviceResult);

//     return 0;
// }

int main() {
    const int N = 1000000;
    double h_arr[N];

    for (int i = 0; i < N; ++i) {
        h_arr[i] = static_cast<float>(i);
    }

    double h_result;

    double* d_arr;
    double* d_result;

    hipMalloc((void**)&d_arr, N * sizeof(double));
    hipMalloc((void**)&d_result, sizeof(double));

    hipMemcpy(d_arr, h_arr, N * sizeof(double), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    kernel<<<1, 1>>>(d_result, d_arr, 0, N - 1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);

    std::cout << "Sum: " << h_result << std::endl;
    std::cout << "Elapsed Time: " << elapsedTime << " ms" << std::endl;

    hipFree(d_arr);
    hipFree(d_result);

    return 0;
}